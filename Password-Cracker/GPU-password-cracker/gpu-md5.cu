
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#define MD5_UNSIGNED_HASH_LEN 16


// C implementation of MD5 from https://rosettacode.org/wiki/MD5#C
typedef union uwb {
    unsigned w;
    unsigned char b[4];
} WBunion;

typedef unsigned Digest[4];

typedef unsigned (*DgstFctn)(unsigned a[]);

__device__ unsigned f0( unsigned abcd[] ){
    return ( abcd[1] & abcd[2]) | (~abcd[1] & abcd[3]);}

__device__ unsigned f1( unsigned abcd[] ){
    return ( abcd[3] & abcd[1]) | (~abcd[3] & abcd[2]);}

__device__ unsigned f2( unsigned abcd[] ){
    return  abcd[1] ^ abcd[2] ^ abcd[3];}

__device__ unsigned f3( unsigned abcd[] ){
    return abcd[2] ^ (abcd[1] |~ abcd[3]);} 

__device__ unsigned *calcKs( unsigned *k) {
    double s, pwr;
    int i;

    pwr = pow(2, 32);
    for (i=0; i<64; i++) {
        double temp = 1 + i;
        s = abs(sin(temp));
        k[i] = (unsigned)( s * pwr );
    }
    return k;
}

// ROtate v Left by amt bits
__device__ unsigned rol( unsigned v, short amt ) {
    unsigned  msk1 = (1<<amt) -1;
    return ((v>>(32-amt)) & msk1) | ((v<<amt) & ~msk1);
}

__device__ void md5(unsigned char *msg, int mlen, uint8_t *hash_code) {
    static Digest h0 = { 0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476 };
//    static Digest h0 = { 0x01234567, 0x89ABCDEF, 0xFEDCBA98, 0x76543210 };
    static DgstFctn ff[] = { &f0, &f1, &f2, &f3 };
    static short M[] = { 1, 5, 3, 7 };
    static short O[] = { 0, 1, 5, 0 };
    static short rot0[] = { 7,12,17,22};
    static short rot1[] = { 5, 9,14,20};
    static short rot2[] = { 4,11,16,23};
    static short rot3[] = { 6,10,15,21};
    static short *rots[] = {rot0, rot1, rot2, rot3 };
    unsigned kspace[64];
    unsigned *k; k = NULL;

    Digest h;
    Digest abcd;
    DgstFctn fctn;
    short m, o, g;
    unsigned f;
    short *rotn;
    union {
        unsigned w[16];
        char     b[64];
    }mm;
    int os = 0;
    int grp, grps, q, p;
    grps  = 1 + (6+8)/64;
    unsigned char msg2[64 + (6+8)];

    if (k==NULL) k= calcKs(kspace);

    for (q=0; q<4; q++) h[q] = h0[q];   // initialize
    {
        memcpy( &(msg2[0]), msg, mlen);
        msg2[mlen] = (unsigned char)0x80;
        q = mlen + 1;
        while (q < 64*grps){ msg2[q] = 0; q++ ; }
        {
//            unsigned char t;
            WBunion u;
            u.w = 8*mlen;
//            t = u.b[0]; u.b[0] = u.b[3]; u.b[3] = t;
//            t = u.b[1]; u.b[1] = u.b[2]; u.b[2] = t;
            q -= 8;
            memcpy(&(msg2[q]), &u.w, 4 );
        }
    }

    for (grp=0; grp<grps; grp++)
    {
        memcpy( mm.b, &(msg2[os]), 64);
        for(q=0;q<4;q++) abcd[q] = h[q];
        for (p = 0; p<4; p++) {
            fctn = ff[p];
            rotn = rots[p];
            m = M[p]; o= O[p];
            for (q=0; q<16; q++) {
                g = (m*q + o) % 16;
                f = abcd[1] + rol( abcd[0]+ fctn(abcd) + k[q+16*p] + mm.w[g], rotn[q%4]);

                abcd[0] = abcd[3];
                abcd[3] = abcd[2];
                abcd[2] = abcd[1];
                abcd[1] = f;
            }
        }
        for (p=0; p<4; p++)
            h[p] += abcd[p];
        os += 64;
    }

    WBunion u;
    int offset;
    int hash_code_offset = 0;
    for (offset=0; offset<4; offset++){
        u.w = h[offset];
        memcpy(&(hash_code[hash_code_offset]), u.b, sizeof(unsigned char) * 4);
        hash_code_offset += 4;
    }
}